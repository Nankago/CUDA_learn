#include "hip/hip_runtime.h"
__global__ void layernorm_forward_kernel3(floatX* __restrict__ out, floatX* __restrict__ mean, floatX* __restrict__ rstd,
                                    const floatX*  __restrict__ inp, const floatX*  __restrict__ weight,
                                    const floatX* __restrict__ bias, int N, int C) {
    const int warp_size = 32;
    int lane_id = threadIdx.x % warp_size;
    int warp_id = threadIdx.x / warp_size;
    int num_warps = blockDim.x / warp_size;

    int idx = blockIdx.x * num_warps + warp_id;
    if(idx >= N) { return; } // guard

    // the row of input that this group of threads is responsible for
    const floatX* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = lane_id; i < C; i += warp_size) {
        sum += (float)x[i];
    }
    sum = warpReduceSum(sum);
    float m = sum / C;
    if(lane_id == 0 && mean != nullptr) {
        __stcs(mean + idx, (floatX)m);
    }

    // rstd
    sum = 0.0f;
    for (int i = lane_id; i < C; i += warp_size) {
        float diff = (float)x[i] - m;
        sum += diff * diff;
    }
    sum = warpReduceSum(sum);
    float s = rsqrtf(sum / C + 1e-5f);
    if(lane_id == 0 && rstd != nullptr) {
        __stcs(rstd + idx, (floatX)s);
    }

    // final normalization and scaling by weight/bias
    floatX* o = out + idx * C;
    for (int c = lane_id; c < C; c += warp_size) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * ((float)__ldcs(x+c) - m);
        __stcs(o+c, (floatX)(n * (float)weight[c] + (float)bias[c]));
    }
}
